
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

void init_array(float*a ,const int N);
void init_mat(float*a ,const int N,const int M);
void print_array(float*a ,const int N);
void print_mat(float*a ,const int N,const int M);


__global__
void kernel(float* vec,float* mat,float* out,const int N,const int M)
{
  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  float sum=0;
  if(tid<M)
  {
    if(tid<M)
       {
          for(int i=0;i<N;i++)
             sum = sum + vec[i]*mat[(i*M)+ tid];
          out[tid]=sum;
       }

  }


}

int main()
{
srand(time(NULL));
 
float *a,*b,*c;
float *d_a,*d_b,*d_c;

int N=3;
int M=4;
//a = [1*3],b= [3*4], c=[1*4]

a = (float*)malloc(sizeof(float)*N);
b = (float*)malloc(sizeof(float)*N*M);
c = (float*)malloc(sizeof(float)*M);

init_array(a,N);
init_mat(b,N,M);
init_array(c,M);

printf("Initial data:\n");
print_array(a,N);

printf("\n\n\n\n");
print_mat(b,N,M);
printf("\n\n\n\n");

print_array(c,M);

printf("\n\n\n\n");

hipMalloc(&d_a,sizeof(float)*N);
hipMalloc(&d_b,sizeof(float)*N*M);
hipMalloc(&d_c,sizeof(float)*M);

hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,sizeof(float)*N*M,hipMemcpyHostToDevice);

kernel<<<M/256+1,256>>>(d_a,d_b,d_c,N,M);

hipError_t err = hipGetLastError();
if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

hipMemcpy(c,d_c,sizeof(float)*M,hipMemcpyDeviceToHost);

printf("Final data:\n");
print_array(c,M);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

delete[] a;
delete[] b;
delete[] c;

return 0;
}

void init_array(float* a,const int N)
{
  for(int i=0;i<N;i++)
    a[i]=rand()%N+1;
}

void init_mat(float*a ,const int N,const int M)
{
   for(int i=0;i<N;i++)
     for(int j=0;j<M;j++)
        a[i*M + j]= rand()%N +1;
}

void print_array(float* a,const int N)
{
  for(int i=0;i<N;i++)
     printf("%f  ",a[i]);
printf("\n");
}

void print_mat(float*a ,const int N,const int M)
{
   for(int i=0;i<N;i++)
   {
     for(int j=0;j<M;j++)
           {
             printf("%  f",a[i*M + j]);
             }
     printf("\n");
}
printf("\n");
}

