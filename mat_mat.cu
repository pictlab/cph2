
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

#define BLOCK_SIZE 16
void init_mat(float*a ,const int N,const int M);
void print_mat(float*a ,const int N,const int M);

__global__
void matrixMultiply(float*a,float*b,float*c,int m,int n,int k)
{
 int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;
 int  sum=0;

 if(col<k && row<m)
 {
    for(int i=0;i<n;i++)
     {
        sum += a[row*n+i]*b[i*k+col];
     }
    c[row*k + col]=sum;

 }


}

int main()
{
srand(time(NULL));

float *a,*b,*c;
float *d_a,*d_b,*d_c;

int M=5;
int N=4;
int P =3;
//a = [5*4],b= [4*3], c=[5*3]

a = (float*)malloc(sizeof(float)*M*N);
b = (float*)malloc(sizeof(float)*N*P);
c = (float*)malloc(sizeof(float)*M*P);


init_mat(a,M,N);
init_mat(b,N,P);
//init_mat(c,M,P);


printf("Initial data:\n");
print_mat(a,M,N);
print_mat(b,N,P);
print_mat(c,M,P);

hipMalloc(&d_a,sizeof(float)*M*N);
hipMalloc(&d_b,sizeof(float)*N*P);
hipMalloc(&d_c,sizeof(float)*M*P);

hipMemcpy(d_a,a,sizeof(float)*M*N,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,sizeof(float)*N*P,hipMemcpyHostToDevice);

//dim3 dimGrid((P + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
dim3 dimGrid(1,1);
dim3 dimBlock(16, 16);
matrixMultiply<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,M,N,P);
hipMemcpy(c,d_c,sizeof(float)*M*P,hipMemcpyDeviceToHost);

printf("Final data:\n");
print_mat(c,M,P);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

delete[] a;
delete[] b;
delete[] c;

return 0;
}

void init_mat(float*a ,const int N,const int M)
{
   for(int i=0;i<N;i++)
     for(int j=0;j<M;j++)
        a[i*M + j]= rand()%N +1;
}

void print_mat(float*a ,const int N,const int M)
{
   for(int i=0;i<N;i++)
   {
     for(int j=0;j<M;j++)
           {
             printf("%  f",a[i*M + j]);
             }
     printf("\n");
}
printf("\n");
}
