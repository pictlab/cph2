
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

void init_array(double *a, const int N);
void print_array(double *a,const int N);
__global__
void vecAdd(double*a,double*b,double*c,int n)
{

 int id = blockIdx.x*blockDim.x + threadIdx.x;

 if(id<n)
   c[id]=a[id]+b[id];

}


int main()
{
 srand(time(NULL));
 int n = 32;

 double *a,*b,*c;
 double *d_a,*d_b,*d_c;
    hipEvent_t start,end;
 const int size = n*sizeof(double);

 a = (double*)malloc(size);
 b = (double*)malloc(size);
 c = (double*)malloc(size);

 hipMalloc(&d_a,size);
 hipMalloc(&d_b,size);
 hipMalloc(&d_c,size);

 init_array(a,n);
 init_array(b,n);
printf("first vector");
 print_array(a,n);
 printf("second vector");
 print_array(b,n);
printf("output");
 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

 int blockSize,gridSize;


 blockSize = 1024;

 gridSize = (int)ceil((float)n/blockSize);
hipEventCreate(&start);
hipEventCreate(&end);
hipEventRecord(start);
 vecAdd<<<gridSize,blockSize>>>(d_a,d_b,d_c,n);
 hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
hipEventRecord(end);
hipEventSynchronize(end);
float time = 0.0;
hipEventElapsedTime(&time,start,end);


 print_array(c,n);
    printf(" Time is :%f ",time);
 hipFree(d_a);
 hipFree(d_b);
 hipFree(d_c);
 delete[] a;
 delete[] b;
 delete[] c;

 return 0;
}

void init_array(double*a,const int N)
{
  for(int i=0;i<N;i++)
     a[i] = rand()%N + 1;


}

void print_array(double*a,const int N)
{
  for(int i=0;i<N;i++)
    printf("%f ",a[i]);

}
