#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

void init_array(double *a, const int N);
void print_array(double *a,const int N);
__global__
void vecAdd(double*a,double*b,double*c,int n)
{
 //get thread id
 int id = blockIdx.x*blockDim.x + threadIdx.x;
 
 if(id<n)
   c[id]=a[id]+b[id];

}


int main()
{
 srand(time(NULL));
 int n = 100;
 
 double *a,*b,*c;
 double *d_a,*d_b,*d_c;

 const int size = n*sizeof(double);

 a = (double*)malloc(size);
 b = (double*)malloc(size);
 c = (double*)malloc(size);

 hipMalloc(&d_a,size);
 hipMalloc(&d_b,size);
 hipMalloc(&d_c,size);

 init_array(a,n);
 init_array(b,n);

 print_array(a,n);
 print_array(b,n);

 hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

 int blockSize,gridSize;
 
 //no. of threads in each thread block
 blockSize = 1024;
 //no of blocks in grid
 gridSize = (int)ceil((float)n/blockSize);

 vecAdd<<<gridSize,blockSize>>>(d_a,d_b,d_c,n);
 
 hipDeviceSynchronize();

 hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

 print_array(c,n);

 hipFree(d_a);
 hipFree(d_b);
 hipFree(d_c);
 delete[] a;
 delete[] b;
 delete[] c;

 return 0;
}

void init_array(double*a,const int N)
{
  for(int i=0;i<N;i++)
     a[i] = rand()%N + 1;


}

void print_array(double*a,const int N)
{
  for(int i=0;i<N;i++)
    printf("%f ",a[i]);
printf("\n");
}
